
#include <hip/hip_runtime.h>
#include <stdio.h>
typedef struct
{
  int *neqn;
} CUSTOMERS;

const int MAX_KERNELS = 72;

__device__ int get_id(void) {
  // Find the ID for this thread, based on which block it is in.
  int idx = threadIdx.x + blockIdx.x * blockDim.x; //thread x coordinate
  int idy = threadIdx.y + blockIdx.y * blockDim.y; //thread y coordinate
  int idz = threadIdx.z + blockIdx.z * blockDim.z; //thread z coordinate

  int size_1d = blockDim.x * gridDim.x;            //n.o. threads on x side
  int size_2d = size_1d * blockDim.y * gridDim.y;  //n.o. thread on x * y side

  return idx + idy * size_1d + idz * size_2d;    //unique id
}

__device__ int get_n(void) {
  return blockDim.x * blockDim.y * blockDim.z * gridDim.x * gridDim.y * gridDim.z;
}

__device__ int id;
__device__ int gpu_array[MAX_KERNELS];

// Device code
__global__ void kernel(CUSTOMERS customers, int *dev_b, int *result) {
  int tid = get_id();

  result[tid] = customers.neqn[tid];//id;
}

const int N = 12;

// Host code
int main(int argc, char const *argv[]) {

  dim3 block_dim(2,1,1); //Number of threads per block
  dim3 grid_dim(2,3,1);  //Number of blocks per grid (cc. 1.2 only supports 2d)
  //Number of kernels:
  int nsize = grid_dim.x * grid_dim.y * grid_dim.z * block_dim.x * block_dim.y * block_dim.z; 

  CUSTOMERS customers;

  // Data on the host and the device, respectively
  int b[nsize], c[nsize]; // host
  int neqn[N];
  int *dev_b , *result;     // device
  int *dev_neqn;

  // Fill the arrays on the host
  for(int i = 0; i < nsize; i++) {
    b[i] = 2;
  }

  for(int i=0;i<N;i++) {
    neqn[i] = i;
  }
  
  // Allocate memory on the device
  hipMalloc((void**)&dev_neqn, sizeof(int) * N);
  hipMalloc((void**)&dev_b, sizeof(int) * nsize);
  hipMalloc((void**)&result, sizeof(int) * nsize);

  // Copy data to the device
  hipMemcpy(dev_neqn, neqn, sizeof(int) * nsize, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeof(int) * nsize, hipMemcpyHostToDevice);

  //Point device pointer in host struct
  customers.neqn = dev_neqn;

  // Launch the kernel with 10 blocks, each with 1 thread
  kernel <<<grid_dim, block_dim>>>(customers, dev_b, result);

  // Copy the result back from the device
  hipMemcpy(c, result, sizeof(int) * nsize, hipMemcpyDeviceToHost);

  // Print the result: "0 5 20 45 80 125 180 245 320 405"
  for(int i = 0; i < nsize; i++) {
    printf("%d ", c[i]);
  }

  printf("\n");

  hipFree(dev_neqn);
  hipFree(dev_b);
  hipFree(result);

  return 0;
}
