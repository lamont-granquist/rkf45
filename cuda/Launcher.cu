#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "Rkf45.hu"
#include "Customers.hu"
#include <time.h>

/*** CUDA ERROR CHECK ***/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


/**************************** HOST ******************************/

//Sorting
int compare(const void *l,const void *r)
{
  CUS* lv = (CUS*)l;
  CUS* rv = (CUS*)r;
  int value = lv->policy - rv->policy;
  if (value == 0)               // Out comment to take away age sorting
    value = lv->age - rv->age;  //
  return value;
}

void sort(CUS *c,int l) {
  qsort(c,l,sizeof(CUS),compare);
}

//Calculate the number of kernels
int get_n_host(dim3 block_dim,dim3 grid_dim) {
  return grid_dim.x * grid_dim.y * grid_dim.z * block_dim.x * block_dim.y * block_dim.z; 
}

// Host code
int main(int argc, char const *argv[]) {
  int n_kernels = 1;
  int gridx = 50;
  int gridy = 50;
    
  if (argc>1) {
      n_kernels = atoi(argv[1]);
  }

  if (argc>2) {
      gridx = atoi(argv[2]);
  }

  if (argc>3) {
      gridy = atoi(argv[3]);
  }

  /********** 0. SETUP **********/
  dim3 block_dim(8,8,5); //Number of threads per block // 320 seems to be best
  dim3 grid_dim(gridx,gridy,1);  //Number of blocks per grid (cc. 1.2 only supports 2d)
  //dim3 block_dim(2,2,1); //Number of threads per block
  //dim3 grid_dim(2,1,1);  //Number of blocks per grid (cc. 1.2 only supports 2d)
  int kernel_size = get_n_host(block_dim,grid_dim);
  int nsize = kernel_size*n_kernels; 

  printf("%i kernels * %i calcs = %i customers\n",n_kernels,kernel_size,nsize);

  /********* -2. GENERATE DATA ***/

  srand(19); //seed
  CUS* cuses = (CUS*)malloc(sizeof(CUS)*nsize);
  for(int i = 0;i < nsize;i++) {
    cuses[i].policy = 1+rand()%6;
    cuses[i].neqn = 1;
    if (cuses[i].policy >= 5) {
      cuses[i].neqn = 2;
    }
    cuses[i].age = 5 + rand()%30;
    cuses[i].end_year = 50;
    cuses[i].start_year = 0;
  }

  /********* -1. SORT DATA *******/
  sort(cuses,nsize);// Out comment to take away sorting

  /********** 1. MALLOC HOST  **********/
  // Data on the host and the device, respectively
  float* result = (float*) malloc(nsize*sizeof(float));

  //Pack
  for(int i = 0;i < nsize;i++) {
    result[i] = 0.0f;
  }

  ///********** 2. MALLOC DEVICE  **********/

  float *dev_result;
  CUS *dev_cuses;
  
  gpuErrchk( hipMalloc((void**)&dev_result, sizeof(float) * nsize));
  gpuErrchk( hipMalloc((void**)&dev_cuses, sizeof(CUS) * nsize));

  /********** 3. COPY HOST TO DEVICE  **********/
  // Copy data to the device
  gpuErrchk( hipMemcpy(dev_cuses, cuses, sizeof(CUS) * nsize, hipMemcpyHostToDevice));

  /********** 4. CUSTOMERS HOLDS POINTERS TO DEVICE **********/
  //Used to hold the pointers

  //********* 5. TIMING START ************/
  //Normal timing
  clock_t start = clock();

  //Cuda timing
  hipEvent_t cuda_start, cuda_stop;
  float cuda_time;
  hipEventCreate(&cuda_start);
  hipEventCreate(&cuda_stop);
  hipEventRecord( cuda_start, 0 );

  int offset = 0;
  /********** 6. LAUNCH WITH CUSTOMERS AND RESULT *********/
  for(int i = 0; i < n_kernels; i++) {
    gpu_kernel <<<grid_dim, block_dim>>>(offset,dev_cuses,dev_result); // GPU
    offset+=kernel_size;
  }
  //test_kernel <<<grid_dim, block_dim>>>(dev_result); // GPU
  ////cpu_kernel(customers,result_cpu); //CPU

  /********** 7. TIMING ENDS *********/
  //Cuda timing
  hipEventRecord( cuda_stop, 0 );
  hipEventSynchronize( cuda_stop );
  hipEventElapsedTime( &cuda_time, cuda_start, cuda_stop );
  hipEventDestroy( cuda_start );
  hipEventDestroy( cuda_stop );
  
  /********** 8. COPY RESULT FROM DEVICE TO HOST *********/
  // Copy the result back from the device
  gpuErrchk( hipMemcpy(result, dev_result, sizeof(float) * nsize, hipMemcpyDeviceToHost));

  /********** 8,5. EXTRA TIMING *********/
  //Normal timing
  clock_t end = clock();
  float time = (float) (end - start) * 1000.0f / CLOCKS_PER_SEC;

  /********** 9. PRINT HOST RESULT  *********/
  // Print the result
  int pa=0;
  for(int i = 0; i < nsize; i++) {
    if (cuses[i].age != pa) {
      printf("%i: %11.7f, policy: %i, age: %i \n",i, result[i],cuses[i].policy,cuses[i].age);
      pa = cuses[i].age;
    }
  }

  /*
  for(int i = 0; i < 51; i++) {
    printf("%i: %.7f\n",i, result_cpu[i]);
  }
  */

  printf("%i kernels * %i calcs = %i customers\n",n_kernels,kernel_size,nsize);
  printf("TIME: %f, CUDA_TIME: %f\n",time,cuda_time);

  /********** 10. FREE MEMORY   *********/
  free(result);
  free(cuses);
  gpuErrchk( hipFree(dev_result));
  gpuErrchk( hipFree(dev_cuses));

  return 0;
}
