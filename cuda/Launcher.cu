#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "Rkf45.hu"
#include "Customers.hu"
#include <time.h>

#include "clireSimulatedIRPaths.hu"

/*** CUDA ERROR CHECK ***/
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/**************************** HOST ******************************/

//Sorting
int compare(const void *l,const void *r)
{
  CUS* lv = (CUS*)l;
  CUS* rv = (CUS*)r;
  int value = lv->policy - rv->policy;
  if (value == 0)               // Out comment to take away age sorting
    value = lv->age - rv->age;  //
  return value;
}

void sort(CUS *c,int l) {
  qsort(c,l,sizeof(CUS),compare);
}

//Calculate the number of kernels
int get_n_host(dim3 block_dim,dim3 grid_dim) {
  return grid_dim.x * grid_dim.y * grid_dim.z * block_dim.x * block_dim.y * block_dim.z; 
}

/* Test of yield curves */
/* De ligger i raekkefolgen:
ir 1 year 50
ir 2 year 50
ir 3 year 50
..
ir 200 year 50
ir 1 year 49
ir 2 year 49
...
ir 200 year 49
ir 1 year 48
*/

// Host code
int main(int argc, char const *argv[]) {

  int n_kernels = 1;
  int gridx = 1;
  int gridy = 1;
  int max_policies = 1;
  int n_yc = 1;
    
  if (argc>1) {
      n_kernels = atoi(argv[1]);
  }

  if (argc>2) {
      gridx = atoi(argv[2]);
  }

  if (argc>3) {
      gridy = atoi(argv[3]);
  }

  if (argc>4) {
      max_policies = atoi(argv[4]);
  }

  if (argc>5) {
      n_yc = atoi(argv[5]);
  }

  /********** 0. SETUP **********/
  dim3 block_dim(8,8,5); //Number of threads per block // 320 seems to be best
  dim3 grid_dim(gridx,gridy,1);  //Number of blocks per grid (cc. 1.2 only supports 2d)
  //dim3 block_dim(2,2,1); //Number of threads per block
  //dim3 grid_dim(2,1,1);  //Number of blocks per grid (cc. 1.2 only supports 2d)
  int kernel_size = get_n_host(block_dim,grid_dim);
  int nsize = kernel_size*n_kernels; 
  int c = float(nsize/n_yc); // number of customers

  printf("%i kernels * %i calcs / %i ir_paths= %i customers\n",n_kernels,kernel_size,n_yc,c);

  /********* -2. GENERATE DATA ***/
  srand(19); //seed

  CUS* cuses = (CUS*)malloc(sizeof(CUS)*c);

  int i=0;
  int id=0;
  while(i < c) {
      int age = 30;//5 + rand()%30; //30
      int end_year = 50;
      int start_year = 0;
      int cj = min(i+max_policies,c);
      for(int j=i;j<cj;j++) {
          cuses[j].id = id;
          cuses[j].age = age;
          cuses[j].end_year = end_year;
          cuses[j].start_year = start_year;

          cuses[j].policy = 1+i%6;//1+rand()%6;
          cuses[j].neqn = 1;
          if (cuses[j].policy >= 5) {
            cuses[j].neqn = 2;
          }
          i++;
      }
      id++;
  }

  //double* collected_results = (float*) malloc(id*sizeof(float));

  /****** GENERATE YIELD CURVES ******/
  double* dev_yieldCurves;
  //generateIRPaths(n_yc,50, &dev_yieldCurves,119); //n_irPaths, years, steps per year, yieldcurve, seed
  double* collected_results = (double*) malloc(id*sizeof(double));


  /********* -1. SORT DATA *******/
  //sort(cuses,c);// sorting

  /********** 1. MALLOC HOST  **********/
  // Data on the host and the device, respectively
  float* result = (float*) malloc(nsize*sizeof(float));

  //Pack
  for(int i = 0;i < nsize;i++) {
    result[i] = 0.0f;
  }

  ///********** 2. MALLOC DEVICE  **********/
  clock_t start1 = clock();

  float *dev_result;
  CUS *dev_cuses;
  
  gpuErrchk( hipMalloc((void**)&dev_result, sizeof(float) * nsize));
  gpuErrchk( hipMalloc((void**)&dev_cuses, sizeof(CUS) * nsize));

  /********** 3. COPY HOST TO DEVICE  **********/
  // Copy data to the device
  gpuErrchk( hipMemcpy(dev_cuses, cuses, sizeof(CUS) * nsize, hipMemcpyHostToDevice));

  /********** 4. CUSTOMERS HOLDS POINTERS TO DEVICE **********/
  //Used to hold the pointers

  //********* 5. TIMING START ************/
  //Normal timing
  clock_t start2 = clock();

  //Cuda timing
  hipEvent_t cuda_start, cuda_stop;
  float cuda_time;
  hipEventCreate(&cuda_start);
  hipEventCreate(&cuda_stop);
  hipEventRecord( cuda_start, 0 );

  int offset = 0;
  /********** 6. LAUNCH WITH CUSTOMERS AND RESULT *********/
  for(int i = 0; i < n_kernels; i++) {
    gpu_kernel <<<grid_dim, block_dim>>>(offset,dev_cuses,dev_result); // GPU
    offset+=kernel_size;
  }

  /********** 7. TIMING ENDS *********/
  //Cuda timing
  hipEventRecord( cuda_stop, 0 );
  hipEventSynchronize( cuda_stop );
  hipEventElapsedTime( &cuda_time, cuda_start, cuda_stop );
  hipEventDestroy( cuda_start );
  hipEventDestroy( cuda_stop );
  
  /********** 8. COPY RESULT FROM DEVICE TO HOST *********/
  clock_t start3 = clock();
  // Copy the result back from the device
  gpuErrchk( hipMemcpy(result, dev_result, sizeof(double) * nsize, hipMemcpyDeviceToHost));

  /********** 8,5. EXTRA TIMING *********/
  //Normal timing

  /*********** COLLECT RESULTS **********/
  //for(int i = 0;i < c;i++)
  //  collected_results[cuses[i].id] += result[i];
  
  for(int i = nsize-10;i < nsize;i++)
    printf("%i: %11.7f \n",i,result[i]);

  clock_t end = clock();
  float time1 = (float) (end - start1) * 1000.0f / CLOCKS_PER_SEC;
  float time2 = (float) (end - start2) * 1000.0f / CLOCKS_PER_SEC;
  float time3 = (float) (end - start3) * 1000.0f / CLOCKS_PER_SEC;
  /********** 9. PRINT HOST RESULT  *********/
  // Print the result
  int pa=0;
  for(int i = 0; i < nsize; i++) {
    if (cuses[i].age != pa) {
      printf("%i: %11.7f, policy: %i, age: %i \n",i, result[i],cuses[i].policy,cuses[i].age);
      pa = cuses[i].age;
    }
  }

  printf("%i kernels * %i calcs = %i customers\n",n_kernels,kernel_size,nsize);
  //printf("TIME: %f, CUDA_TIME: %f\n",time,cuda_time);
  printf("TIME1: %f\n",time1);
  printf("TIME2: %f\n",time2);
  printf("TIME3: %f\n",time3);

  /********** 10. FREE MEMORY   *********/
  free(result);
  free(cuses);
  gpuErrchk( hipFree(dev_result));
  gpuErrchk( hipFree(dev_cuses));

  hipDeviceReset();
  return 0;
}
