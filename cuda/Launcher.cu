#include <stdio.h>
#include "Rkf45.hu"
#include "Customers.hu"

/**************************** HOST ******************************/

//Calculate the number of kernels
int get_n_host(dim3 block_dim,dim3 grid_dim) {
  return grid_dim.x * grid_dim.y * grid_dim.z * block_dim.x * block_dim.y * block_dim.z; 
}

// Host code
int main(int argc, char const *argv[]) {

  dim3 block_dim(2,2,3); //Number of threads per block
  dim3 grid_dim(2,3,1);  //Number of blocks per grid (cc. 1.2 only supports 2d)
  int nsize = get_n_host(block_dim,grid_dim); 

  // Data on the host and the device, respectively
  int result[nsize];
  int *dev_result;
  CUSTOMERS *dev_customers; 
  CUSTOMERS *customers;

  customers = (CUSTOMERS*) malloc(sizeof(CUSTOMERS)*nsize);

  for(int i = 0;i < nsize;i++) {
    customers[i].neqn = 1;
    customers[i].policy = 1;
    customers[i].end_year = 40;
  }

  // Allocate memory on the device
  hipMalloc((void**)&dev_customers, sizeof(CUSTOMERS) * nsize);
  hipMalloc((void**)&dev_result, sizeof(int) * nsize);

  // Copy data to the device
  hipMemcpy(dev_customers, customers, sizeof(CUSTOMERS) * nsize, hipMemcpyHostToDevice);

  // Launch the kernel with 10 blocks, each with 1 thread
  //kernel <<<grid_dim, block_dim>>>(dev_customers,dev_result);
  test_kernel <<<grid_dim, block_dim>>>(dev_customers, dev_result);

  // Copy the result back from the device
  hipMemcpy(result, dev_result, sizeof(int) * nsize, hipMemcpyDeviceToHost);

  // Print the result: "0 5 20 45 80 125 180 245 320 405"
  for(int i = 0; i < nsize; i++) {
    printf("%d ", result[i]);
  }

  printf("\n");

  hipFree(dev_result);

  return 0;
}
