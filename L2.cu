//Main

#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char const *argv[]) {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  int device;
  for (device = 0; device < deviceCount; ++device) {
      hipDeviceProp_t deviceProp;
      hipGetDeviceProperties(&deviceProp, device);
      printf("Device %d has L2 cache size %d.\n",
             device, deviceProp.l2CacheSize);
      printf("Number of multiprocessors:     %d\n",  deviceProp.multiProcessorCount);
  }
}
