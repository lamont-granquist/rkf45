
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int get_id(void) {
  // Find the ID for this thread, based on which block it is in.
  int idx = threadIdx.x + blockIdx.x * blockDim.x; //thread x coordinate
  int idy = threadIdx.y + blockIdx.y * blockDim.y; //thread y coordinate
  int idz = threadIdx.z + blockIdx.z * blockDim.z; //thread z coordinate

  int size_1d = blockDim.x * gridDim.x;            //n.o. threads on x side
  int size_2d = size_1d * blockDim.y * gridDim.y;  //n.o. thread on x * y side

  return idx + idy * size_1d + idz * size_2d;    //unique id
}

__device__ int get_n(void) {
  return blockDim.x * blockDim.y * blockDim.z * gridDim.x * gridDim.y * gridDim.z;
}

//
__device__ int deviceFunction3(int* id,int* variable1,int* variable2,int* variable3,int* variable4) {
  *variable1 += 8;
  *variable4 += 7;
  *variable2 += 9;
  *variable3 += 2;

  return *variable1 + *variable2 + *variable3;
}

__device__ int deviceFunction2(int* id,int* variable1,int* variable2,int* variable3,int* variable4) {
  *variable3 += 8; 
  *variable1 += deviceFunction3(id,variable1,variable2,variable3,variable4);
  *variable4 += deviceFunction3(id,variable1,variable2,variable3,variable4);

  return *variable3 + *variable4;
}

__device__ int deviceFunction1(int* id,int* variable1,int* variable2,int* variable3,int* variable4) {
  *variable1 += *id;
  *variable4 += 2;
  *variable2 += deviceFunction2(id,variable1,variable2,variable3,variable4);
  *variable3 += *variable2 + *variable4;
  return *variable1 + *variable2 + *variable3 + *variable4;
}

// Kernel
__global__ void kernel(int *dev_a, int *dev_b, int *dev_c) {
  int id = get_id();
  int variable1 = 3;
  int variable2 = 5;
  int variable3 = 8;
  int variable4 = 8;

  dev_c[id] = deviceFunction1(&id,&variable1,&variable2,&variable3,&variable4);
}

// Host code
int main(int argc, char const *argv[]) {

  dim3 block_dim(2,1,1); //Number of threads per block
  dim3 grid_dim(2,3,1);  //Number of blocks per grid (cc. 1.2 only supports 2d)
  //Number of kernels:
  int nsize = grid_dim.x * grid_dim.y * grid_dim.z * block_dim.x * block_dim.y * block_dim.z; 

  // Data on the host and the device, respectively
  int a[nsize], b[nsize], c[nsize]; // host
  int *dev_a , *dev_b , *dev_c;     // device

  // Fill the arrays on the host
  for(int i = 0; i < nsize; i++) {
    a[i] = i * i;
    b[i] = 2;
  }

  // Allocate memory on the device
  hipMalloc((void**)&dev_a, sizeof(int) * nsize);
  hipMalloc((void**)&dev_b, sizeof(int) * nsize);
  hipMalloc((void**)&dev_c, sizeof(int) * nsize);

  // Copy data to the device
  hipMemcpy(dev_a, a, sizeof(int) * nsize, hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, sizeof(int) * nsize, hipMemcpyHostToDevice);

  // Launch the kernel with 10 blocks, each with 1 thread
  kernel <<<grid_dim, block_dim>>>(dev_a, dev_b, dev_c);

  // Copy the result back from the device
  hipMemcpy(c, dev_c, sizeof(int) * nsize, hipMemcpyDeviceToHost);

  // Print the result: "0 5 20 45 80 125 180 245 320 405"
  for(int i = 0; i < nsize; i++) {
    printf("%d ", c[i]);
  }

  printf("\n");

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}
